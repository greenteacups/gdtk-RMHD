#include "hip/hip_runtime.h"
// cell.cu
// Include file for chicken.
// PJ 2022-09-11

#ifndef CELL_INCLUDED
#define CELL_INCLUDED

#include <string>
#include <sstream>
#include <stdexcept>

#include "number.cu"
#include "vector3.cu"
#include "gas.cu"
#include "flow.cu"
#include "face.cu"

using namespace std;

namespace Face {
    // Symbolic names for the faces of the cell and of the block.
    constexpr int iminus = 0;
    constexpr int iplus = 1;
    constexpr int jminus = 2;
    constexpr int jplus = 3;
    constexpr int kminus = 4;
    constexpr int kplus = 5;

    array<string,6> names {"iminus", "iplus", "jminus", "jplus", "kminus", "kplus"};
};

int Face_indx_from_name(string name)
{
    if (name == "iminus") return Face::iminus;
    if (name == "iplus") return Face::iplus;
    if (name == "jminus") return Face::jminus;
    if (name == "jplus") return Face::jplus;
    if (name == "kminus") return Face::kminus;
    if (name == "kplus") return Face::kplus;
    throw runtime_error("Invalid face name: " + name);
}


namespace SourceTerms {
    array<string,3> names{"none", "manufactured_solution"};
    //
    constexpr int none = 0;
    constexpr int manufactured_solution = 1;
};

int source_terms_from_name(string name)
{
    if (name == "none") return SourceTerms::none;
    if (name == "manufactured_solution") return SourceTerms::manufactured_solution;
    return SourceTerms::none;
}


namespace IOvar {
    // Following the new IO model for Eilmer, we set up the accessor functions
    // for the flow data that is held in the flow data files.
    // These accessor functions are associated with the Cell structure.

    // Keep the following list consistent with the GlobalConfig.iovar_names list
    // in chkn_prep.py and with the symbolic constants just below.
    vector<string> names {"posx", "posy", "posz", "vol",
                              "p", "T", "rho", "e", "YB", "a",
                              "velx", "vely", "velz"};

    // We will use these symbols to select the varaible of interest.
    constexpr int posx = 0;
    constexpr int posy = posx + 1;
    constexpr int posz = posy + 1;
    constexpr int vol = posz + 1;
    constexpr int p = vol + 1;
    constexpr int T = p + 1;
    constexpr int rho = T + 1;
    constexpr int e = rho + 1;
    constexpr int YB = e + 1;
    constexpr int a = YB + 1;
    constexpr int velx = a + 1;
    constexpr int vely = velx + 1;
    constexpr int velz = vely + 1;
    constexpr int n = velz + 1; // number of symbols that point to the flow variables
}


struct FVCell {
    Vector3 pos; // position of centroid
    number volume;
    number iLength, jLength, kLength; // These lengths are used in the interpolation fns.
    FlowState fs;
    // We will keep connections to the pieces compising the cell as indices
    // into the block's arrays.
    // Although we probably don't need build and keep this data for the structured grid,
    // it simplifies some of the geometry and update code and may ease the use of
    // unstructured grids at a later date.
    array<int,8> vtx{0, 0, 0, 0, 0, 0, 0, 0};
    array<int,6> face{0, 0, 0, 0, 0, 0};

    string toString() const {
        ostringstream repr;
        repr << "Cell(pos=" << pos.toString() << ", volume=" << volume;
        repr << ", iLength=" << iLength << ", jLength=" << jLength << ", kLength=" << kLength;
        repr << ", fs=" << fs.toString();
        repr << ", vtx=["; for(auto v : vtx) repr << v << ","; repr << "]";
        repr << ", face=["; for(auto v : face) repr << v << ","; repr << "]";
        repr << ")";
        return repr.str();
    }

    void iovar_set(int i, number val)
    {
        switch (i) {
        case IOvar::posx: pos.x = val; break;
        case IOvar::posy: pos.y = val; break;
        case IOvar::posz: pos.z = val; break;
        case IOvar::vol: volume = val; break;
        case IOvar::p: fs.gas.p = val; break;
        case IOvar::T: fs.gas.T = val; break;
        case IOvar::rho: fs.gas.rho = val; break;
        case IOvar::e: fs.gas.e = val; break;
        case IOvar::YB: fs.gas.YB = val; break;
        case IOvar::a: fs.gas.a = val; break;
        case IOvar::velx: fs.vel.x = val; break;
        case IOvar::vely: fs.vel.y = val; break;
        case IOvar::velz: fs.vel.z = val; break;
        default:
            throw runtime_error("Invalid selection for IOvar: "+to_string(i));
        }
    }

    number iovar_get(int i)
    {
        switch (i) {
        case IOvar::posx: return pos.x;
        case IOvar::posy: return pos.y;
        case IOvar::posz: return pos.z;
        case IOvar::vol: return volume;
        case IOvar::p: return fs.gas.p;
        case IOvar::T: return fs.gas.T;
        case IOvar::rho: return fs.gas.rho;
        case IOvar::e: return fs.gas.e;
        case IOvar::YB: return fs.gas.YB;
        case IOvar::a: return fs.gas.a;
        case IOvar::velx: return fs.vel.x;
        case IOvar::vely: return fs.vel.y;
        case IOvar::velz: return fs.vel.z;
        default:
            throw runtime_error("Invalid selection for IOvar: "+to_string(i));
        }
        // So we never return from here.
    }

    __host__ __device__
    number estimate_local_dt(Vector3 inorm, Vector3 jnorm, Vector3 knorm, number cfl)
    {
        // We assume that the cells are (roughly) hexagonal and work with
        // velocities normal to the faces.
        number isignal = iLength/(fabs(fs.vel.dot(inorm))+fs.gas.a);
        number jsignal = jLength/(fabs(fs.vel.dot(jnorm))+fs.gas.a);
        number ksignal = kLength/(fabs(fs.vel.dot(knorm))+fs.gas.a);
        return cfl * fmin(fmin(isignal,jsignal),ksignal);
    }

    __host__ __device__
    void add_source_terms(ConservedQuantities& dUdt, int isrc)
    {
        switch (isrc) {
        case SourceTerms::none:
            break;
        case SourceTerms::manufactured_solution:
            dUdt[CQI::mass] += zero; // [TODO] implement the actual calculation.
            dUdt[CQI::xMom] += zero;
            dUdt[CQI::yMom] += zero;
            dUdt[CQI::zMom] += zero;
            dUdt[CQI::totEnergy] += zero;
            dUdt[CQI::YB] += zero;
            break;
        default:
            break;
        }
        return;
    }

    __host__ __device__
    void eval_dUdt(ConservedQuantities& dUdt, FVFace faces[], int isrc)
    // These are the spatial (RHS) terms in the semi-discrete governing equations.
    {
        number vol_inv = one/volume;
        auto& fim = faces[face[Face::iminus]];
        auto& fip = faces[face[Face::iplus]];
        auto& fjm = faces[face[Face::jminus]];
        auto& fjp = faces[face[Face::jplus]];
        auto& fkm = faces[face[Face::kminus]];
        auto& fkp = faces[face[Face::kplus]];
        // Introducing local variables for the data helps
        // promote coalesced global memory access on the GPU.
        number area_im = fim.area; ConservedQuantities F_im = fim.F;
        number area_ip = fip.area; ConservedQuantities F_ip = fip.F;
        number area_jm = fjm.area; ConservedQuantities F_jm = fjm.F;
        number area_jp = fjp.area; ConservedQuantities F_jp = fjp.F;
        number area_km = fkm.area; ConservedQuantities F_km = fkm.F;
        number area_kp = fkp.area; ConservedQuantities F_kp = fkp.F;
        //
        for (int i=0; i < CQI::n; i++) {
            // Integrate the fluxes across the interfaces that bound the cell.
            number surface_integral = area_im*F_im[i] - area_ip*F_ip[i]
                + area_jm*F_jm[i] - area_jp*F_jp[i] + area_km*F_km[i] - area_kp*F_kp[i];
            // Then evaluate the derivatives of conserved quantity.
            // Note that conserved quantities are stored per-unit-volume.
            dUdt[i] = vol_inv*surface_integral;
        }
        //
        if (isrc != SourceTerms::none) add_source_terms(dUdt, isrc);
        return;
    } // end eval_dUdt()

}; // end Cell


__host__
ostream& operator<<(ostream& os, const FVCell c)
{
    os << c.toString();
    return os;
}

#endif
